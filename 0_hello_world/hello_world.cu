
#include <hip/hip_runtime.h>
#include<stdio.h>
#include <iostream>
#include <unistd.h>
__global__ void hello_world(void)
{
  printf("GPU: Hello world!\n");
  // std::cout << "Hello GPU" << std::endl;
}
int main(int argc,char **argv)
{
  printf("CPU: Hello world!\n");
  std::cout << "Hello CPU" << std::endl;
  hello_world<<<2,10>>>();
  sleep(5);
  std::cout << "sleep over " << std::endl; 
  hipDeviceReset();//if no this line ,it can not output hello world from gpu 将当前GPU状态恢复到初始状态
  return 0;
}
